
#include <hip/hip_runtime.h>
#include <cstdint>
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <vector>

void random_init(float *data, size_t size) {
    for (size_t i = 0; i < size; ++i) {
        data[i] = float(rand()) / RAND_MAX;
    }
}

__global__ void transposeNative(float *input, float *output, int m, int n) {
    int colID_input = threadIdx.x + blockDim.x * blockIdx.x;
    int rowID_input = threadIdx.y + blockDim.y * blockIdx.y;
    if (rowID_input < m && colID_input < n) {
        int index_input = colID_input + rowID_input * n;
        int index_output = rowID_input + colID_input * m;
        output[index_output] = input[index_input];
    }
}


int main() {
    int m = 8192;
    int n = 4096;
    float *h_input, *h_output;
    hipHostMalloc(&h_input, m * n * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_output, m * n * sizeof(float), hipHostMallocDefault);
    random_init(h_input, m * n);
    float *d_input, *d_output;
    hipMalloc(&d_input,  m * n * sizeof(float));
    hipMalloc(&d_output,  m * n * sizeof(float));

    hipMemcpy(d_input, h_input,  m * n * sizeof(float), hipMemcpyDefault);
    
    dim3 block(8, 32);
    dim3 grid((n + block.x -1)/block.x, (m + block.y-1)/block.y);
    transposeNative<<<grid, block>>>(d_input, d_output, m , n);

    hipMemcpy(h_output, d_output,  m * n * sizeof(float), hipMemcpyDefault);

    hipDeviceSynchronize();
    const hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf(hipGetErrorString(error));
        return -1;
    } else {
        printf("success \n");
    }
    return 0;
}